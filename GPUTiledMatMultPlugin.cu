
#include "GPUTiledMatMultPlugin.h"

void GPUTiledMatMultPlugin::input(std::string infile) {
   readParameterFile(infile);
}

void GPUTiledMatMultPlugin::run() {}

void GPUTiledMatMultPlugin::output(std::string outfile) {
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

int M, N, P;
 M = atoi(myParameters["M"].c_str());
 N = atoi(myParameters["N"].c_str());
 P = atoi(myParameters["P"].c_str());
 numARows = M;
 numAColumns = N;
 numBRows = N;
 numBColumns = P;
 numCRows = M;
 numCColumns = P;

  hostA = (float*) malloc (M*N*sizeof(float));
  hostB = (float*) malloc (N*P*sizeof(float));
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numARows * numBColumns * sizeof(float));

  numCRows    = numARows;
  numCColumns = numBColumns;

 std::ifstream myinput((std::string(PluginManager::prefix())+myParameters["matrix1"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M*N; ++i) {
        float k;
        myinput >> k;
        hostA[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+myParameters["matrix2"]).c_str(), std::ios::in);
 for (i = 0; i < N*P; ++i) {
        float k;
        myinput2 >> k;
        hostB[i] = k;
 }



  //@@ Allocate GPU memory here
  hipMalloc(&deviceA, sizeof(float) * numARows * numAColumns);
  hipMalloc(&deviceB, sizeof(float) * numBRows * numBColumns);
  hipMalloc(&deviceC, sizeof(float) * numCRows * numCColumns);
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns,
             hipMemcpyHostToDevice);


  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid((numCColumns - 1) / TILE_WIDTH + 1,
               (numCRows - 1) / TILE_WIDTH + 1, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  //@@ Launch the GPU Kernel here
  matrixMultiply<<<dimGrid, dimBlock>>>(
      deviceA, deviceB, deviceC, numARows, numAColumns, numBRows,
      numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns,
             hipMemcpyDeviceToHost);

  std::ofstream outsfile(outfile.c_str(), std::ios::out);

        for (i = 0; i < M*P; ++i){
                outsfile << hostC[i];//std::setprecision(0) << a[i*N+j];
                outsfile << "\n";
        }


  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);


  free(hostA);
  free(hostB);
  free(hostC);

}

PluginProxy<GPUTiledMatMultPlugin> GPUTiledMatMultPluginProxy = PluginProxy<GPUTiledMatMultPlugin>("GPUTiledMatMult", PluginManager::getInstance());
